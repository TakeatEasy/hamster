#include "hip/hip_runtime.h"
#include "NumCpp.hpp"
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
 
__global__ void add_vectors(int *d_a) {
    d_a[threadIdx.x] += 1;
}

int main()
{
    auto a = nc::random::randInt<int>({10, 10}, 0, 100);
    std::cout << a;

    int* d_a;
    hipMalloc(&d_a, sizeof(int) * 100);
    hipMemcpy(d_a, a.data(), 100*sizeof(int), hipMemcpyHostToDevice);

    add_vectors<<<1, 100>>>(d_a);
    hipMemcpy(a.data(), d_a, 100*sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "After kernel\n";
    std::cout << a;
    hipFree(d_a);
 
    return EXIT_SUCCESS;
}
